
#include <hip/hip_runtime.h>
#include <cstdio>

int main(void) {
    fprintf(stdout, "Hello world\n");

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    } else {
        printf("Found %d GPUs\n");
    }

}